
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define N 100000
__global__ void prime_count(int *countd)
{
        int i = blockIdx.x*blockDim.x+threadIdx.x;
        int flag = 0;
        int j;
        if((i>2)&&(i<N))
        {
                for(j=2;j<i;j++)
                {
                if(i%j == 0)
                {
                flag =1;
                break;
                }
                }
        if(flag == 0)
        {
                countd[i]=1;
        }
        else
        {
        countd[i]=0;
        }
        }
}


int main()
{
        int i,j;
        int count,flag,*cnt,*cntd;
        double exe_time;
        struct timeval stop_time,start_time;

        count = 1;
        gettimeofday(&start_time,NULL);
        cnt=(int *)malloc(N*sizeof(int));
        hipMalloc(&cntd,N*sizeof(int));

        int num_threads_per_block = 256;
        int total_threads = N;
        int num_blocks = total_threads / num_threads_per_block + 1;
        prime_count<<<num_blocks,num_threads_per_block>>>(cntd);
        hipMemcpy(cnt,cntd,N*sizeof(int),hipMemcpyDeviceToHost);
for(i=3;i<N;i++)
{
        if(cnt[i]==1)
        {
                count++;
        }
}

gettimeofday(&stop_time,NULL);
exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));

printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n",count,exe_time);
free(cnt);
hipFree(cntd);

}
